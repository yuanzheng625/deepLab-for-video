#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  caffe_gpu_scal(count, Dtype(1 / sqrt(count)), diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  switch (this->layer_param_.euclidean_loss_param().type()) {
  case EuclideanLossParameter_Type_L2:
    loss_ = 0.5 * dot;
    break;
  case EuclideanLossParameter_Type_L2sqrt:
    loss_ = sqrt(dot);
    break;
  default:
    LOG(FATAL) << "Unknown Type";
  }
  top[0]->mutable_cpu_data()[0] = loss_;
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int count = bottom[0]->count();
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      Dtype alpha;
      switch (this->layer_param_.euclidean_loss_param().type()) {
      case EuclideanLossParameter_Type_L2:
	alpha = sign * top[0]->cpu_diff()[0] / sqrt(count);
	break;
      case EuclideanLossParameter_Type_L2sqrt:
	alpha = sign * top[0]->cpu_diff()[0] / sqrt(count) /
	  std::max(loss_, Dtype(1e-6));
	break;
      default:
	LOG(FATAL) << "Unknown Type";
      }
      caffe_gpu_axpby(
          count,                              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
